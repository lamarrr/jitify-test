#include "hip/hip_runtime.h"
#include "rmm/cuda_stream_view.hpp"

// Problems:
// - atomic header not found
// - some headers contain CPU-code that are not executed on the device, i.e. <stdexcept> used in RMM_CUDA_TRY in rmm/error.hpp
//

__global__ void add_kernel(int const *__restrict x, int const *__restrict y,
                           int *__restrict z, unsigned long long int length)
{
  if (threadIdx.x < length)
  {
    z[threadIdx.x] = x[threadIdx.x] + y[threadIdx.x];
  }
}
